/*
	Noel Lopes is an Assistant Professor at the Polytechnic Institute of Guarda, Portugal (for more information see readme.txt)
    Copyright (C) 1999, 2000, 2001, 2002, 2003, 2004, 2005, 2006, 2007, 2008, 2009, 2010 Noel de Jesus Mendon�a Lopes

	This file is part of Multiple Back-Propagation.

    Multiple Back-Propagation is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "hip/hip_runtime.h"

#if __DEVICE_EMULATION__

    Cuda::Cuda() {
        numberDevices = 1;
        device = 0;
    }
    
#else

    Cuda::Cuda() {
		device = 0;
	    if (hipGetDeviceCount(&numberDevices) != hipSuccess) numberDevices = 0;
	    
	    for(; device < numberDevices; device++) {
		    if(hipGetDeviceProperties(&deviceProperties, device) == hipSuccess && deviceProperties.major >= 1 && deviceProperties.major != 9999) { // 9999 - Emulation
		        if (hipSetDevice(device) == hipSuccess) break;
		    }
	    }
    }

#endif