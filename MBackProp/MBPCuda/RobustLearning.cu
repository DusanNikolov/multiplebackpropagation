#include "hip/hip_runtime.h"
/*
	Noel Lopes is an Assistant Professor at the Polytechnic Institute of Guarda, Portugal (for more information see readme.txt)
    Copyright (C) 1999, 2000, 2001, 2002, 2003, 2004, 2005, 2006, 2007, 2008, 2009, 2010 Noel de Jesus Mendon�a Lopes

	This file is part of Multiple Back-Propagation.

    Multiple Back-Propagation is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "MBPkernels.h"

KERNEL RobustLearning(CUDA_FLOATING_TYPE * rmsF, CUDA_FLOATING_TYPE * bestRMS, CUDA_FLOATING_TYPE maxErrorGrowth, int layers, int * numberWeights, CUDA_FLOATING_TYPE ** weights, CUDA_FLOATING_TYPE ** bestWeights, CUDA_FLOATING_TYPE ** learningRate, CUDA_FLOATING_TYPE r, CUDA_FLOATING_TYPE ** lastDeltaWithoutLearningMomentum, CUDA_FLOATING_TYPE ** lastDelta) {
    __shared__ CUDA_FLOATING_TYPE rms;
    __shared__ CUDA_FLOATING_TYPE bRMS;
    
    rms = *rmsF;
    bRMS = *bestRMS;
    
    if (rms < bRMS) {
        for (int l = 0; l < layers; l++) {
            if (threadIdx.x < numberWeights[l]) bestWeights[l][threadIdx.x] = weights[l][threadIdx.x];
        }
        
        if (threadIdx.x == 0) *bestRMS = rms;
    } else if (rms >= bRMS * maxErrorGrowth) {
        for (int l = 0; l < layers; l++) {
            if (threadIdx.x < numberWeights[l]) {
                weights[l][threadIdx.x] = bestWeights[l][threadIdx.x];
                
                learningRate[l][threadIdx.x] *= r;
                
                lastDeltaWithoutLearningMomentum[l][threadIdx.x] = CUDA_VALUE(0.0);
                lastDelta[l][threadIdx.x] = CUDA_VALUE(0.0);
            }
        }
    }
}